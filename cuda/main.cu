#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <chrono>
#include <vector>
#include <cstring>
#include "hip/hip_runtime.h"


using namespace std;

const int MAX_V = 1000;          // Maximum number of vertices
const int MAX_EDGES = 10;        // Maximum number of edges per vertex
const int MAX_GRAPHS = 100000;   // Maximum number of graphs
const int THREADS_PER_BLOCK = 128; // Number of threads per block

class Graph {
public:
    int V; // Number of vertices
    int* adj; // Flattened adjacency lists
    int* degree; // Number of edges per vertex
};

__device__ bool isSubgraphDevice(int V, int* adj, int* degree, int sub_V, int* sub_adj, int* sub_degree, int sub_root, int start_vertex) {
    int to_visit_queue[2 * MAX_V];
    int mapping_keys[MAX_V];
    int mapping_values[MAX_V];
    bool visited[MAX_V] = { false };
    int to_visit_start = 0;
    int to_visit_end = 0;
    int mapping_size = 0;

    to_visit_queue[to_visit_end++] = start_vertex;
    to_visit_queue[to_visit_end++] = sub_root;
    mapping_keys[mapping_size] = sub_root;
    mapping_values[mapping_size++] = start_vertex;
    visited[start_vertex] = true;

    while (to_visit_start < to_visit_end) {
        int curr_v = to_visit_queue[to_visit_start++];
        int curr_sub_v = to_visit_queue[to_visit_start++];

        for (int i = 0; i < sub_degree[curr_sub_v]; ++i) {
            int sub_w = sub_adj[curr_sub_v * MAX_EDGES + i];
            bool matched = false;
            for (int j = 0; j < degree[curr_v]; ++j) {
                int w = adj[curr_v * MAX_EDGES + j];
                int k;
                for (k = 0; k < mapping_size; ++k) {
                    if (mapping_keys[k] == sub_w) {
                        if (mapping_values[k] == w) {
                            matched = true;
                        }
                        break;
                    }
                }
                if (k == mapping_size) {
                    mapping_keys[mapping_size] = sub_w;
                    mapping_values[mapping_size++] = w;
                    to_visit_queue[to_visit_end++] = w;
                    to_visit_queue[to_visit_end++] = sub_w;
                    matched = true;
                    visited[w] = true;
                    break;
                }
            }
            if (!matched) {
                return false;
            }
        }
    }
    return true;
}

__global__ void processGraphsKernel(Graph* graphs, int* results, int count, int* sub_adj, int* sub_degree, int sub_V, int sub_root) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < count) {
        Graph g = graphs[index];
        int occurrences = 0;

        for (int i = 0; i < g.V; ++i) {
            if (g.degree[i] >= sub_degree[sub_root] && isSubgraphDevice(g.V, g.adj, g.degree, sub_V, sub_adj, sub_degree, sub_root, i)) {
                occurrences++;
            }
        }

        results[index] = occurrences;
    }
}

void loadGraphs(const string& filename, Graph* graphs[], int& count) {
    ifstream file(filename);
    string line;
    count = 0;

    while (getline(file, line) && count < MAX_GRAPHS) {
        stringstream ss(line);
        int u, v;
        vector<pair<int, int>> edges;
        int max_vertex = -1;

        while (ss >> u >> v) {
            edges.push_back({u, v});
            max_vertex = max(max_vertex, max(u, v));
        }

        Graph* g = new Graph();
        g->V = max_vertex + 1;
        g->degree = new int[g->V]();
        g->adj = new int[g->V * MAX_EDGES]();

        for (const auto& edge : edges) {
            int from = edge.first;
            int to = edge.second;
            g->adj[from * MAX_EDGES + g->degree[from]++] = to;
        }

        // cout << "Loaded graph with " << g->V << " vertices." << endl;
        // for (int i = 0; i < g->V; ++i) {
        //     cout << "Vertex " << i << " degree: " << g->degree[i] << endl;
        // }

        graphs[count++] = g;
    }
}

void prepareAndCopyGraphsToGPU(Graph** hostGraphs, Graph** deviceGraphs, int graphCount) {
    hipMalloc(deviceGraphs, graphCount * sizeof(Graph));

    for (int i = 0; i < graphCount; i++) {
        Graph g = *hostGraphs[i];
        hipMalloc(&(g.adj), g.V * MAX_EDGES * sizeof(int));
        hipMemcpy(g.adj, hostGraphs[i]->adj, g.V * MAX_EDGES * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&(g.degree), g.V * sizeof(int));
        hipMemcpy(g.degree, hostGraphs[i]->degree, g.V * sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(&((*deviceGraphs)[i]), &g, sizeof(Graph), hipMemcpyHostToDevice);
        // cout << "Copied graph " << i + 1 << " to GPU." << endl;
    }
}

int findRoot(int* degree, int V) {
    int max_degree = 0;
    int root = -1;
    for (int i = 0; i < V; ++i) {
        if (degree[i] > max_degree) {
            max_degree = degree[i];
            root = i;
        }
    }
    return root;
}

int main() {
    const string inputFileName = "data.txt";
    Graph* hostGraphs[MAX_GRAPHS];
    int graphCount = 0;
    loadGraphs(inputFileName, hostGraphs, graphCount);

    Graph* deviceGraphs;
    prepareAndCopyGraphsToGPU(hostGraphs, &deviceGraphs, graphCount);

    // Define the subgraph
    int sub_V = 8;
    int sub_degree[MAX_V] = {0};
    int sub_adj[MAX_V * MAX_EDGES] = {0};

    vector<pair<int, int>> sub_edges = {
        {3, 4}, {4, 5}, {3, 2}, {2, 1}, {3, 6}, {6, 7}
    };

    for (const auto& edge : sub_edges) {
        int from = edge.first;
        int to = edge.second;
        sub_adj[from * MAX_EDGES + sub_degree[from]++] = to;
    }

    int sub_root = findRoot(sub_degree, sub_V);

    int* device_sub_adj;
    int* device_sub_degree;

    hipMalloc((void**)&device_sub_adj, sub_V * MAX_EDGES * sizeof(int));
    hipMemcpy(device_sub_adj, sub_adj, sub_V * MAX_EDGES * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&device_sub_degree, sub_V * sizeof(int));
    hipMemcpy(device_sub_degree, sub_degree, sub_V * sizeof(int), hipMemcpyHostToDevice);

    int* results = new int[graphCount];
    int* deviceResults;
    hipMalloc((void**)&deviceResults, graphCount * sizeof(int));

    dim3 blocks((graphCount + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 threads(THREADS_PER_BLOCK);

    auto start = chrono::high_resolution_clock::now();
    processGraphsKernel<<<blocks, threads>>>(deviceGraphs, deviceResults, graphCount, device_sub_adj, device_sub_degree, sub_V, sub_root);
    auto finish = chrono::high_resolution_clock::now();

    hipMemcpy(results, deviceResults, graphCount * sizeof(int), hipMemcpyDeviceToHost);
    ofstream file("results.txt");
    // cout << "Total graphs: " << graphCount << endl;
    for (int i = 0; i < graphCount; i++) {
        // cout << "Graph " << (i + 1) << " occurrences: " << results[i] << endl;
        file << "Found " << results[i] << " occurrences of subgraph in main graph " << (i + 1) << "." << endl;
    }

    auto durationMs = chrono::duration_cast<chrono::microseconds>(finish - start);
    std::cout << "Program time: " << durationMs.count() << " microseconds." << std::endl;

    hipFree(deviceGraphs);
    hipFree(deviceResults);
    hipFree(device_sub_adj);
    hipFree(device_sub_degree);
    delete[] results;
    for (int i = 0; i < graphCount; i++) {
        delete[] hostGraphs[i]->adj;
        delete[] hostGraphs[i]->degree;
        delete hostGraphs[i];
    }

    return 0;
}
